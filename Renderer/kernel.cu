#include "hip/hip_runtime.h"
#include "kernel.cuh"

#include "hip/hip_runtime.h"
#include ""
#include "cutil_math.h"
#include "hiprand/hiprand_kernel.h"


#include <cmath>
#include <stdio.h>
#include <stdexcept>
#include <sstream>

#define cudaCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "CUDA ERROR: %s %s %d\n", hipGetErrorString(code), file, line);
		std::stringstream ss;
		ss << "CUDA ERROR: " << hipGetErrorString(code) << " " << file << " " << line << "\n";
		if (abort) throw std::runtime_error(ss.str());
	}
}

__device__ float
leakyReLU(float val)
{
	return fmaxf(0, val) + 0.1 * fminf(0, val);
}

__device__ void
matvecmul(bool relu, int M, int K, const float* a, const float* b, const float* c, float* d)
{
	for (int i = 0; i < M; ++i)
	{
		float val = 0;
		for (int j = 0; j < K; ++j)
		{
			val += a[i * K + j] * b[j];
		}
		val += c[i];
		d[i] = (relu) ? leakyReLU(val) : val;
	}
}

float __device__ computeSDF(const float3& pos, float * buffer, const
	Renderer::Parameters& params)
{
	float3 val = make_float3(1, 0, 0);

	int M = params.N, K = 3;

	bool relu = true;

	float* weights = params.weights, * biases = params.biases;

	buffer[0] = pos.x;
	buffer[1] = pos.y;
	buffer[2] = pos.z;
	float* output_buffer = buffer;
	for (int l = 0; l < params.H + 1; ++l)
	{
		float* input_buffer = buffer + (M * (l % 2));
		output_buffer = buffer + (M * ((l + 1) % 2));
		if (l == params.H)
		{
			M = 1;
			relu = false;
		}

		matvecmul(relu, M, K, weights, input_buffer, biases, output_buffer);

		weights += M * K;
		biases += M;

		K = M;
	}

	float output = std::tanh(output_buffer[0]);
	return output;
}

float3 __device__ objectColor(const float3& pos, const Renderer::Parameters& params)
{
	return make_float3(1.0f);
}

float __device__ distFromOrigin(const float3& position, const float3& direction)
{
	float3 n = normalize(direction);
	float dist = dot(-1 * position, n) / dot(n, n);
	float3 p = position + dist * n;
	return length(p);
}

float __device__ distToSphere(const float3& position, const float3& direction)
{
	float a = dot(direction, direction);
	float b = 2.0 * dot(position, direction);
	float c = dot(position, position) - 1;
	float discriminant = b * b - 4 * a * c;
	if (discriminant < 0.0)
	{
		return -1;
	}
	else
	{
		float numerator = -b - sqrt(discriminant);
		if (numerator > 0)
		{
			return numerator / (2.0 * a);
		}

		numerator = -b + sqrt(discriminant);
		if (numerator > 0)
		{
			return numerator / (2.0 * a);
		}
		else
		{
			return -1;
		}
	}
}

#define EPS 0.000001

__device__ float3 rayMarching(const float3& position, const float3& direction, const Renderer::Parameters& params)
{
	extern __shared__ float shared_mem[];
	size_t offset = 2 * params.N * (threadIdx.x + threadIdx.y * blockDim.x);
	float* buffer = shared_mem + offset;
	float3 color = params.background_color;

	float3 pos = position;
	float3 dir = normalize(direction);
	float intersect_dist = distToSphere(pos, dir);

	//For this renderer, all points occupy the unit sphere, so nothing outside needs to be rendered.
	if (intersect_dist < 0)
	{
		return color;
	}
	else
	{
		pos += (intersect_dist + EPS) * dir;
	}

	float pos_len = length(pos);
	while (pos_len < 1)
	{

		float dist = computeSDF(pos, buffer, params);

		if (dist < params.min_dist)
		{
			float nx = (computeSDF(make_float3(pos.x + params.eps, pos.y, pos.z), buffer, params) - computeSDF(make_float3(pos.x - params.eps, pos.y, pos.z), buffer, params));
			float ny = (computeSDF(make_float3(pos.x, pos.y + params.eps, pos.z), buffer, params) - computeSDF(make_float3(pos.x, pos.y - params.eps, pos.z), buffer, params));
			float nz = (computeSDF(make_float3(pos.x, pos.y, pos.z + params.eps), buffer, params) - computeSDF(make_float3(pos.x, pos.y, pos.z - params.eps), buffer, params));
			float3 normal = normalize(make_float3(nx, ny, nz));

			//Diffuse lighting
			float3 light_vec = pos - params.light.positionf();
			float light_dot_normal = dot(light_vec, normal) / length(light_vec);
			float diff_angle = std::acosf(light_dot_normal);
			float diff_scale = fmaxf(fminf(1.0f - (fabs(diff_angle - M_PI) / M_PI), 1), 0);
			diff_scale *= params.light.diffuseStrength();

			//Specular lighting
			float3 reflected = light_vec - 2 * light_dot_normal * normal;
			float3 cam_vec = pos - params.cam.positionf();
			float spec_angle = std::acosf(dot(cam_vec, reflected) / (length(reflected) * length(cam_vec)));
			float spec_scale = pow(fmaxf(fminf(1.0f - (fabs(spec_angle - M_PI) / M_PI), 1), 0), (float)params.light.getSpecularPower());
			spec_scale *= params.light.specularStrength();

			color = objectColor(pos, params) *(diff_scale + spec_scale) + params.light.ambientStrength() * params.light.colorf();
			return color;
		}

		pos += dist * dir;
		pos_len = length(pos);
	}

	return color;
}


__global__ void renderImage(Renderer::Parameters params)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x >= params.width || y >= params.height) return;

	float px = (x / float(params.width) - 0.5f) * 2.0f;
	float py = -(y / float(params.height) - 0.5f) * 2.0f * float(params.height) / float(params.width);
	float3 direction = normalize(params.cam.sidef() * px + params.cam.upf() * py + params.cam.directionf() * params.cam.fovScale());
	float3 color = rayMarching(params.cam.positionf(), direction, params);

	params.device_image[3 * (x + y * params.width) + 0] = fmaxf(fminf(255 * color.x, 255), 0);
	params.device_image[3 * (x + y * params.width) + 1] = fmaxf(fminf(255 * color.y, 255), 0);
	params.device_image[3 * (x + y * params.width) + 2] = fmaxf(fminf(255 * color.z, 255),0);
}

unsigned char*
Renderer::makeImage(unsigned int width, unsigned int height)
{
	unsigned char* deviceImage;
	cudaCheck(hipMalloc(&deviceImage, 3 * width * height));
	return deviceImage;
}

void
Renderer::gpuDelete(unsigned char* image)
{
	cudaCheck(hipFree(image));
}


void
Renderer::gpuDelete(float* data)
{
	cudaCheck(hipFree(data));
}

void
Renderer::render()
{
	dim3 block_size((params_.N <= 64) ? 8 : 4, 8);
	dim3 grid_size(params_.width / block_size.x + 1, params_.height / block_size.y + 1);
	renderImage<<<grid_size, block_size, 2 * block_size.x * block_size.y * params_.N * sizeof(float)>>>(params_);
	cudaCheck(hipPeekAtLastError());
	cudaCheck(hipMemcpy(params_.image, params_.device_image, 3 * params_.width * params_.height, hipMemcpyDeviceToHost));
}

void
Renderer::copyDataToGPU(float** dst, const float* src, size_t numel)
{
	cudaCheck(hipMalloc(dst, sizeof(float) * numel));
	cudaCheck(hipMemcpy(*dst, src, sizeof(float) * numel, hipMemcpyHostToDevice));
}
